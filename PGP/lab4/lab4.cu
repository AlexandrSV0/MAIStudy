#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <math.h>

using namespace std;

#define FastIO ios_base::sync_with_stdio(false); cin.tie(nullptr), cout.tie(nullptr); 
#define CATCH(call)                                                            \
    do {                                                                                 \
        hipError_t res = call;                                                 \
        if (res != hipSuccess)                                   			   \
        {                                                          					        \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n",  \
                    __FILE__, __LINE__, hipGetErrorString(res)); 	 \
            exit(0);                                              			   			  \
        }                                                        							 \
    } while (0);

struct comparator {												
    __host__ __device__ bool operator()(double a, double b) { 
        return abs(a) < abs(b); 
    } 
}; 

__host__ void readMatrixAndB(double* matrix, int n) {
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            std::cin >> matrix[j*n + i];
    
    for (int i = 0; i < n; i++)
        std::cin >> matrix[n*n + i];
}

__global__ void swapRows(double* data, int n, int curRow, int rowToSwap) {
     int id_x = blockDim.x*blockIdx.x + threadIdx.x; 
    for (int j = id_x; j < n+1; j += blockDim.x*gridDim.x) { 
        double cp = data[j*n + curRow]; 
        data[j*n + curRow] = data[j*n+ rowToSwap]; 
        data[j*n + rowToSwap] = cp; 
    }
}

__global__ void Gauss(double* data, int n, int row) { 
    int id_x = blockDim.x*blockIdx.x + threadIdx.x; 
    int id_y = blockDim.y*blockIdx.y + threadIdx.y; 
    int off_x = blockDim.x*gridDim.x; 
    int off_y = blockDim.y*gridDim.y; 
    
    int column = row;
    for (int i = id_x + row + 1; i < n; i+= off_x)
        for (int j = id_y + column+1; j < n+1; j+= off_y)
            data[j*n + i] += data[j*n + row] * (-data[column*n + i] / data[column*n + row]); 
}

__host__ void solveEquatation(double* matrix, double* res, int n) {
    for (int i = n-1; i >= 0; i--) { 
        double b_i = matrix[n*n + i];
        for (int j = n - 1; j > i; j--)
            b_i -= res[j] * matrix[j*n + i];
        res[i] = b_i / matrix[i*n + i];
    }
}

int main() {
    FastIO
    int n;
    cin >> n;
    int matrixSize = n*n + n;
    double *matrix = new double[matrixSize];
    readMatrixAndB(matrix, n);

    double* dataDev; 
    CATCH(hipMalloc(&dataDev, sizeof(double)*matrixSize));
    CATCH(hipMemcpy(dataDev, matrix, sizeof(double)*matrixSize, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CATCH(hipEventCreate(&start));
    CATCH(hipEventCreate(&stop));
    CATCH(hipEventRecord(start));
    comparator comparator; 
    for (int j = 0; j < n-1; j++) { 
        //ptr на столбец j 
        auto columnPtr = thrust::device_pointer_cast(dataDev + j*n);	 
        // ptr на max el (в столбце по строкам от i до n) 
        int i = j; 
        auto maxPtr = thrust::max_element(columnPtr + i,  columnPtr + n, comparator);	
        int maxPos = maxPtr - columnPtr; 
        if (maxPos != i) swapRows<<<256, 256>>>(dataDev, n, i, maxPos); 

        Gauss<<<dim3(64, 64), dim3(32, 32)>>>(dataDev, n, i); 
    }

    CATCH(hipEventRecord(stop));
    CATCH(hipEventSynchronize(stop));

    float t;
    CATCH(hipEventElapsedTime(&t, start, stop));
    CATCH(hipEventDestroy(start));
    CATCH(hipEventDestroy(stop));

    printf("time = %f ms\n", t);

    CATCH(hipMemcpy(matrix, dataDev, sizeof(double)*matrixSize, hipMemcpyDeviceToHost));
    CATCH(hipFree(dataDev)); 

    double* res = new double[n];
    solveEquatation(matrix, res, n);
  
    // for (int i = 0; i < n; i++)
    //     printf("%10e ", res[i]);

    free(matrix); 	
    free(res); 
}