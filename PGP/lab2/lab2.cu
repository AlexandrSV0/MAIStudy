
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>


#define CATCH_ERR(call) \
do { \
	hipError_t res = call;	\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n", \
				__FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while(0) \


typedef struct {
    int w;
    int h;
} ImageSize;


__global__ void ssaa_smoothing(hipTextureObject_t tex, uchar4 *dev, const ImageSize out_img, int w_diff, int h_diff) {
    int pixelBlock = w_diff*h_diff;
    int id_x = blockDim.x*blockIdx.x + threadIdx.x;
    int id_y = blockDim.y*blockIdx.y + threadIdx.y;
    int offset_x = blockDim.x*gridDim.x;
    int offset_y = blockDim.y*gridDim.y;

    for (int x = id_x; x < out_img.w; x += offset_x) {
        for (int y = id_y; y < out_img.h; y += offset_y) {
            int3 block_data;
            block_data.x = 0;
            block_data.y = 0;
            block_data.z = 0;
            int alpha;

            for (int i = 0; i < w_diff; i++) {
                for (int j = 0; j < h_diff; j++) {
                    uchar4 pix = tex2D<uchar4>(tex,  x*w_diff + i, y*h_diff + j);
                    block_data.x += pix.x;
                    block_data.y += pix.y;
                    block_data.z += pix.z;
                    alpha = pix.w;
                }
            }

            block_data.x /= pixelBlock;
            block_data.y /= pixelBlock;
            block_data.z /= pixelBlock;
            dev[y*out_img.w + x] = make_uchar4(block_data.x, block_data.y, block_data.z, alpha);
        }
    }
}


int main() {
	ImageSize out_img;
	int w, h;
	std::string file_in, file_out;
	std::cin >> file_in >> file_out;
	std::cin >> out_img.w >> out_img.h;
    
	std::ifstream fs_in(file_in, std::ios::in | std::ios::binary);
	fs_in.read((char *)&w, sizeof(w));
	fs_in.read((char *)&h, sizeof(h));
	int img_size = w*h;
	int out_img_size = out_img.w*out_img.h;
	uchar4 *data = new uchar4[img_size];
	fs_in.read((char *)data, w*h*sizeof(data[0]));
    fs_in.close();
	
	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CATCH_ERR(hipMallocArray(&arr, &ch, w, h));
	CATCH_ERR(hipMemcpy2DToArray(arr, 0, 0, data, w*sizeof(uchar4), w*sizeof(uchar4), h, hipMemcpyHostToDevice));

	struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = false;

    hipTextureObject_t tex = 0;
    CATCH_ERR(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

	uchar4 *dev;
	CATCH_ERR(hipMalloc(&dev, sizeof(uchar4)*out_img_size));
	
    hipEvent_t start, stop;  
    CATCH_ERR(hipEventCreate(&start));
    CATCH_ERR(hipEventCreate(&stop));
    CATCH_ERR(hipEventRecord(start));

    ssaa_smoothing<<<dim3(64, 64), dim3(32, 32)>>>(tex, dev, out_img, w/out_img.w, h/out_img.h);

    CATCH_ERR(hipDeviceSynchronize());
    CATCH_ERR(hipGetLastError());

    CATCH_ERR(hipEventRecord(stop));
    CATCH_ERR(hipEventSynchronize(stop));
    float t;
    CATCH_ERR(hipEventElapsedTime(&t, start, stop));
    CATCH_ERR(hipEventDestroy(start));
    CATCH_ERR(hipEventDestroy(stop));
    printf("time = %f ms\n", t);

	CATCH_ERR(hipGetLastError());
    CATCH_ERR(hipMemcpy(data, dev, sizeof(uchar4)*out_img_size, hipMemcpyDeviceToHost));

	std::ofstream fs_out(file_out, std::ios::out | std::ios::binary);
	fs_out.write((char *)&out_img.w, sizeof(out_img.w));
	fs_out.write((char *)&out_img.h, sizeof(out_img.h));
	fs_out.write((char *)data, out_img_size*sizeof(data[0])); 
	fs_out.close();

	CATCH_ERR(hipDestroyTextureObject(tex));
	CATCH_ERR(hipFreeArray(arr));
	CATCH_ERR(hipFree(dev));
	free(data);
	return 0;    
}