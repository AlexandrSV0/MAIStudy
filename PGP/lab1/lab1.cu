
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CSC(call) 							\
do { 										\
	hipError_t status = call;				\
	if (status != hipSuccess) {																				\
		fprintf(stderr, "ERROR is %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));		\
		exit(0);								\
	}											\
} while(0)

__global__ void findElByElMaximums(double* vec1, double* vec2, double* res, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    while (i < n) {
        res[i] = vec1[i] > vec2[i] ? vec1[i] : vec2[i];
        i += offset;
    }
}

int main() {
    int n;
    scanf("%d", &n);

    double *vec1 = (double *) malloc(sizeof(double) * n);
    double *vec2 = (double *) malloc(sizeof(double) * n);
    double *res = (double *) malloc(sizeof(double) * n);

    for (int i = 0; i < n; i++) {
        scanf("%lf", &vec1[i]);
    }
    
    for (int i = 0; i < n; i++) {
        scanf("%lf", &vec2[i]);
    }

    double *dev1, *dev2, *dev_res;
    hipMalloc(&dev1, sizeof(double) * n);
    hipMalloc(&dev2, sizeof(double) * n);
    hipMalloc(&dev_res, sizeof(double) * n);
    hipMemcpy(dev1, vec1, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(dev2, vec2, sizeof(double) * n, hipMemcpyHostToDevice);


    hipEvent_t start, stop;  
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&stop));
    CSC(hipEventRecord(start));

    findElByElMaximums<<<1024, 1024>>>(dev1, dev2, dev_res, n);

    CSC(hipDeviceSynchronize());
    CSC(hipGetLastError());

    CSC(hipEventRecord(stop));
    CSC(hipEventSynchronize(stop));
    float t;
    CSC(hipEventElapsedTime(&t, start, stop));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(stop));
	
    printf("time = %f ms\n", t);

    hipMemcpy(res, dev_res, sizeof(double) * n, hipMemcpyDeviceToHost);

    // for (int i = 0; i < n; i++) {
    //     printf("%.10e ", res[i]);
    // }
    
    hipFree(dev1);
    hipFree(dev2);
    hipFree(dev_res);
    free(vec1);
    free(vec2);
    free(res);
    return 0;
}