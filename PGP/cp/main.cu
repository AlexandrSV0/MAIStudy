
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>

#define CSC(call)                                                                                                                                       \
do {                                                                                                                                                            \
    hipError_t status = call;                                                                                                                       \
    if (status != hipSuccess) {                                                                                                                    \
        fprintf(stderr, "ERROR is %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));  \
        exit(0);                                                                                                                                                \
    }                                                                                                                                                             \
} while(0)

const int POLYGONS_CNT = 26;
const int RGB_CONST = 255;
const int BUFF_SIZE = 256;
const double EPSILON = 1e-10;

const std::string DEFAULT_INPUT = R"""(126
output/img_%d.data
720 720 100
7.0 3.0 0.0		2.0 1.0     2.0 6.0 1.0     0.0 0.0
2.0 0.0 0.0     0.5 0.1     1.0 4.0 1.0     0.0 0.0
0 -3 0.3	0 0 1	    1.7
1 0.5 1		0.3 0.7		0.8     1
-1 3 0.5    1 0 0      1.5
-5 -5 -1    -5.0 5.0 -1.0    5.0 5.0 -1.0    5.0 -5.0 -1.0  0.0 1.0 0.0
-10.0 0.0 20.0     0.3 0.2 0.1
4
)""";


struct Figure {
    double3 center;
    double3 color;
    double radius;

    __host__ __device__ Figure() {}
    
    friend std::istream& operator>>(std::istream& is, Figure& f) {
        is >> f.center.x >> f.center.y >> f.center.z;
        is >> f.color.x >> f.color.y >> f.color.z;
        is >> f.radius;
        return is;
    }
};

struct Motion_params {
    double r0, z0, phi0, A, Az, wr, wz, wphi, pr, pz;
    __host__ __device__ Motion_params() {}
    
    friend std::istream& operator>>(std::istream& is, Motion_params& params) {
        is >> params.r0 >> params.z0 >> params.phi0 >> params.A >> params.Az >> params.wr >> params.wz >> params.wphi >> params.pr >> params.pz;
        return is;
    }

};

__host__ __device__ double dot(double3 a, double3 b) {    
    return a.x*b.x + a.y*b.y + a.z*b.z;
}

__host__ __device__ double3 diff(double3 a, double3 b) {
	return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__host__ __device__ double3 sum(double3 a, double3 b) {
	return {a.x + b.x, a.y + b.y, a.z + b.z};
}

__host__ __device__ uint4 sum(uint4 data, uchar4 cur_pixel) {
    data.x += cur_pixel.x;
    data.y += cur_pixel.y;
    data.z += cur_pixel.z;
    return data;
}

__host__ __device__ uint4 divide(uint4 data, int num) {
    data.x /= num;
    data.y /= num;
    data.z /= num;
    return data;
}

__host__ __device__ double3 prod(double3 a, double3 b) {   
    return make_double3(
        a.y*b.z - a.z*b.y,
        a.z*b.x - a.x*b.z,
        a.x*b.y - a.y*b.x
    );
}

__host__ __device__ double3 normalize(double3 v) {
    double l = sqrt(dot(v, v));
    return make_double3(v.x / l, v.y / l, v.z / l);
}

__host__ __device__ double3 mult(double3 a, double3 b, double3 c, double3 v) {
    return make_double3(
        a.x*v.x + b.x*v.y + c.x*v.z,
        a.y*v.x + b.y*v.y + c.y*v.z,
        a.z*v.x + b.z*v.y + c.z*v.z
    );
}

__host__ __device__ double3 mult(double3 a, double num) {
    return make_double3(a.x*num, a.y*num, a.z*num);
}


struct Polygon {
    double3 a;
    double3 b;
    double3 c;
    uchar4 color;

    __host__ __device__ Polygon() {}
    __host__ __device__ Polygon(double3 a, double3 b, double3 c, uchar4 color) : a(a), b(b), c(c), color(color) {}
};

__host__ __device__ uchar4 ray(Polygon *polygons, double3 pos, double3 dir, double3 lpos, uchar4 lcol) {
    const auto empty_uchar4 = make_uchar4(0, 0, 0, RGB_CONST);
    int k = -1, k_min = -1;
    double ts_min;
    while (++k < POLYGONS_CNT) {
        double3 e1 = diff(polygons[k].b, polygons[k].a);
        double3 e2 = diff(polygons[k].c, polygons[k].a);
        double3 p = prod(dir, e2);
        double div = dot(p, e1);

        if (fabs(div) < EPSILON) {
            continue;
        }

        double3 t = diff(pos, polygons[k].a);
        double u = dot(p, t) / div;
        if (u < 0.0 || u > 1.0) {
            continue;
        }

        double3 q = prod(t, e1);
        double v = dot(q, dir) / div;
        if (v < 0.0 || v + u > 1.0) {
            continue;
        }

        double ts = dot(q, e2) / div; 
        if (ts < 0.0) {
            continue;
        }
        
        if (k_min == -1 || ts < ts_min) {
            k_min = k;
            ts_min = ts;
        }
    }

    if (k_min == -1) {
        return empty_uchar4;
    }

    pos = sum(mult(dir, ts_min), pos);
    dir = diff(lpos, pos);
    double length = sqrt(dot(dir, dir));
    dir = normalize(dir);
    k = -1;
    while (++k < POLYGONS_CNT) { // we have to process the same way the light source
        double3 e1 = diff(polygons[k].b, polygons[k].a);
        double3 e2 = diff(polygons[k].c, polygons[k].a);
        double3 p = prod(dir, e2);
        double div = dot(p, e1);

        if (fabs(div) < EPSILON) {
            continue;
        }
        
        double3 t = diff(pos, polygons[k].a);
        double u = dot(p, t) / div;
        if (u < 0.0 || u > 1.0) {
            continue;
        }

        double3 q = prod(t, e1);
        double v = dot(q, dir) / div;
        if (v < 0.0 || v + u > 1.0) {
            continue;
        }

        double ts = dot(q, e2) / div; 
        if (ts > 0.0 && ts < length && k != k_min) {
            return empty_uchar4;
        }
    }

    uchar4 k_color = polygons[k_min].color;
    double x = k_color.x * lcol.x;
    double y = k_color.y * lcol.y;
    double z = k_color.z * lcol.z;
    return make_uchar4(x, y, z, RGB_CONST);
}

// CPU version
__host__ __device__ void make_cpu_render(uchar4 *data, Polygon *polygons, double3 cam_pos, double3 cam_view, int w, int h, double angle, double3 lpos, uchar4 lcol) {
    double dw = 2.0 / (w - 1.0);
    double dh = 2.0 / (h - 1.0);
    double z = 1.0 / tan(angle * M_PI / 360.0);

    double3 bz = normalize(diff(cam_view, cam_pos));
    double3 bx = normalize(prod(bz, {0.0, 0.0, 1.0}));
    double3 by = normalize(prod(bx, bz));

    int i = -1, j = -1;
    while (++i < w) {
        while (++j < h) {
            double3 v = make_double3(-1.0 + dw*i, (-1.0 + dh*j)*h / w, z);
            double3 dir = mult(bx, by, bz, v);
            data[(h - 1 - j) * w + i] = ray(polygons, cam_pos, normalize(dir), lpos, lcol);
        }
        j = -1;
    }
}

// GPU version
__global__ void kernel_render(uchar4 *data, Polygon *polygons, double3 cam_pos, double3 cam_view, int w, int h, double angle, double3 lpos, uchar4 lcol) {
    int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    int id_y = blockDim.y * blockIdx.y + threadIdx.y;
    int offset_x = blockDim.x * gridDim.x;
    int offset_y = blockDim.y * gridDim.y;

    double dw = 2.0 / (w - 1.0);
    double dh = 2.0 / (h - 1.0);
    double z = 1.0 / tan(angle * M_PI / 360.0);

    double3 bz = normalize(diff(cam_view, cam_pos));
    double3 bx = normalize(prod(bz, {0.0, 0.0, 1.0}));
    double3 by = normalize(prod(bx, bz));

    for (int i = id_x; i < w; i += offset_x) {
        for (int j = id_y; j < h; j += offset_y) {
            double3 v = make_double3(-1.0 + dw * i, (-1.0 + dh * j) * h / w, z);
            double3 dir = mult(bx, by, bz, v);
            data[(h - 1 - j) * w + i] = ray(polygons, cam_pos, normalize(dir), lpos, lcol);
        }
    }
}

__host__ __device__ uint4 sum_pixels(uchar4 *data, int w, int h, int x, int y, int sqrtSamples) {
    auto block_data = make_uint4(0, 0, 0, 0);
    for (int i = 0; i < sqrtSamples; ++i)
        for (int j = 0; j < sqrtSamples; ++j)
            block_data = sum(block_data, data[w*sqrtSamples * (y*sqrtSamples + j) + (x*sqrtSamples + i)]);
    return block_data;
}

// CPU version
__host__ __device__ void make_cpu_ssaa(uchar4 *data, uchar4 *data_out, int w, int h, int sqrtSamples) {
    int pixelBlock = sqrtSamples*sqrtSamples;
    for (int x = 0; x < w; x++) {
        for (int y = 0; y < h; y++) {
            auto block_data = sum_pixels(data, w, h, x, y, sqrtSamples);
            block_data = divide(block_data, pixelBlock);
            data_out[y*w + x] = make_uchar4(block_data.x, block_data.y, block_data.z, RGB_CONST);
        }
    }
}

// GPU version
__global__ void kernel_ssaa(uchar4 *data, uchar4 *data_out, int w, int h, int sqrtSamples) {
    int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    int id_y = blockDim.y * blockIdx.y + threadIdx.y;
    int offset_x = blockDim.x * gridDim.x;
    int offset_y = blockDim.y * gridDim.y;

    int pixelBlock = sqrtSamples * sqrtSamples;
    for (int x = id_x; x < w; x += offset_x) {
        for (int y = id_y; y < h; y += offset_y) { 
            auto block_data = sum_pixels(data, w, h, x, y, sqrtSamples);
            block_data = divide(block_data, pixelBlock);
            data_out[y * w + x] = make_uchar4(block_data.x, block_data.y, block_data.z, RGB_CONST);
        }
    }
}

// building scene and figure objects
void build_space(Polygon *polygons, double3 *floor, uchar4 color, int &start) {
    polygons[start] = Polygon(floor[0], floor[1], floor[2], color);
    polygons[++start] = Polygon(floor[0], floor[2], floor[3], color);
}

void build_tetrahedron(Polygon *polygons, double3 center, uchar4 color, double r, int &start) {
    double a = r * sqrt(3);
    double half_a = a / 2;
    double quarter_a = a / 4;

    auto p1 = make_double3(center.x - half_a, center.y, center.z - quarter_a);
    auto p2 = make_double3(center.x, center.y + r, center.z - quarter_a);
    auto p3 = make_double3(center.x + half_a, center.y, center.z - quarter_a);
    auto p4 = make_double3(center.x, center.y, center.z + r);

    polygons[++start] = Polygon(p1, p2, p3, color);
    polygons[++start] = Polygon(p1, p2, p4, color);
    polygons[++start] = Polygon(p1, p3, p4, color);
    polygons[++start] = Polygon(p2, p3, p4, color);
}

void build_hexahedron(Polygon *polygons, double3 center, uchar4 color, double r, int &start) {
    double a = 3 * r / sqrt(3);
    
    auto p0 = make_double3(center.x - a / 2, center.y - a / 2, center.z - a / 2);
    auto p1 = make_double3(p0.x, p0.y, p0.z);
    auto p2 = make_double3(p0.x, p0.y + a, p0.z);
    auto p3 = make_double3(p0.x + a, p0.y + a, p0.z);
    auto p4 = make_double3(p0.x + a, p0.y, p0.z);
    auto p5 = make_double3(p0.x, p0.y, p0.z + a);
    auto p6 = make_double3(p0.x, p0.y + a, p0.z + a);
    auto p7 = make_double3(p0.x + a, p0.y + a, p0.z + a);
    auto p8 = make_double3(p0.x + a, p0.y, p0.z + a);

    polygons[++start] = Polygon(p1, p2, p3, color);
    polygons[++start] = Polygon(p3, p4, p1, color);
    polygons[++start] = Polygon(p7, p8, p4, color);
    polygons[++start] = Polygon(p4, p3, p7, color);
    polygons[++start] = Polygon(p3, p2, p6, color);
    polygons[++start] = Polygon(p6, p7, p3, color);
    polygons[++start] = Polygon(p5, p6, p2, color);
    polygons[++start] = Polygon(p2, p1, p5, color);
    polygons[++start] = Polygon(p4, p8, p5, color);
    polygons[++start] = Polygon(p5, p1, p4, color);
    polygons[++start] = Polygon(p7, p6, p5, color);
    polygons[++start] = Polygon(p5, p8, p7, color);
}

void build_octahedron(Polygon *polygons, double3 center, uchar4 color, double r, int &start) {
    auto p1 = make_double3(center.x, center.y - r, center.z);
    auto p2 = make_double3(center.x - r, center.y, center.z);
    auto p3 = make_double3(center.x, center.y + r, center.z);
    auto p4 = make_double3(center.x + r, center.y, center.z);
    auto p5 = make_double3(center.x, center.y, center.z - r);
    auto p6 = make_double3(center.x, center.y, center.z + r);


    polygons[++start] = Polygon(p1, p2, p5, color);
    polygons[++start] = Polygon(p2, p3, p5, color);
    polygons[++start] = Polygon(p3, p4, p5, color);
    polygons[++start] = Polygon(p4, p1, p5, color);
    polygons[++start] = Polygon(p1, p6, p2, color);
    polygons[++start] = Polygon(p2, p6, p3, color);
    polygons[++start] = Polygon(p3, p6, p4, color);
    polygons[++start] = Polygon(p4, p6, p1, color);
}

double3 read_double3() {
    double3 num;
    std::cin >> num.x >> num.y >> num.z;
    return num;
}

void writeToFile(char *output_dir, int w, int h, int screen_size, uchar4 *data_out) {
	std::ofstream outputFS(output_dir, std::ios::out | std::ios::binary);  
	outputFS.write((char *) &w, sizeof(int));   
	outputFS.write((char *) &h, sizeof(int)); 
	outputFS.write((char *) data_out, screen_size*sizeof(uchar4));  
	outputFS.close();
}

void print_log(int k, float time, long long cnt) {
    std::cout << k << '\t' << time << '\t' << cnt << '\n';
}

void print_default() {
    std::cout << DEFAULT_INPUT;
}

/*
run with:
    '--cpu' to make rendering without using GPU;
    '--default' to get default input;
    '--gpu' \or nothing\  make rendering with using GPU;
*/
int main(int argc, char *argv[]) {
    bool GPU = true;
    if (argc >= 2) {
        auto key = std::string(argv[1]);
        if (key == "--default") {
            print_default();
            return 0;
        }
        GPU = key == "--cpu" ? false : true; 
    }

    int frames; 
    std::cin >> frames;
    
    char output_dir[BUFF_SIZE];
    std::cin >> output_dir;

    int w, h;
    std::cin >> w >> h;
    
    double angle;
    std::cin >> angle;

    Motion_params cam, dir;
    std::cin >> cam >> dir;

    Figure tetrahedron, hexahedron, octahedron;
    std::cin >> tetrahedron >> hexahedron >> octahedron;
    
    double3 floor_coord[4];
    for (int i = 0; i < 4; i++)
        floor_coord[i] = read_double3();

    auto floor_color = read_double3();

    auto light_coord = read_double3();
    auto light_color = read_double3();
    
    double sqrtSamples;
    std::cin >> sqrtSamples;

    auto floor_color_uch4 = make_uchar4(floor_color.x * RGB_CONST, floor_color.y * RGB_CONST, floor_color.z * RGB_CONST, RGB_CONST);
    auto tetrahedron_color = make_uchar4(tetrahedron.color.x * RGB_CONST, tetrahedron.color.y * RGB_CONST, tetrahedron.color.z * RGB_CONST, RGB_CONST);
    auto hexahedron_color = make_uchar4(hexahedron.color.x * RGB_CONST, hexahedron.color.y * RGB_CONST, hexahedron.color.z * RGB_CONST, RGB_CONST);
    auto octahedron_color = make_uchar4(octahedron.color.x * RGB_CONST, octahedron.color.y * RGB_CONST, octahedron.color.z * RGB_CONST, RGB_CONST);

    int start = 0;
    Polygon polygons[POLYGONS_CNT];
    build_space(polygons, floor_coord, floor_color_uch4, start);
    build_tetrahedron(polygons, tetrahedron.center, tetrahedron_color, tetrahedron.radius, start);
    build_hexahedron(polygons, hexahedron.center, hexahedron_color, hexahedron.radius, start);
	build_octahedron(polygons, octahedron.center, octahedron_color, octahedron.radius, start);

    auto lpos = make_double3(light_coord.x, light_coord.y, light_coord.z);
    auto lcol = make_uchar4(light_color.x*RGB_CONST, light_color.y*RGB_CONST, light_color.z*RGB_CONST, RGB_CONST);

    const int screen_size = w*h;
    uchar4 *dev, *dev_out,
        *data_out = (uchar4*) malloc(sizeof(uchar4) * screen_size),
        *data = (uchar4*) malloc(sizeof(uchar4) * screen_size * sqrtSamples * sqrtSamples);
    
    Polygon *dev_polygons;
    
    if (GPU) {
        CSC(hipMalloc(&dev, sizeof(uchar4)*screen_size*sqrtSamples*sqrtSamples));
        CSC(hipMalloc(&dev_out, sizeof(uchar4)*screen_size));
        CSC(hipMalloc(&dev_polygons, sizeof(Polygon)*POLYGONS_CNT));
        CSC(hipMemcpy(dev_polygons, polygons, sizeof(Polygon)*POLYGONS_CNT, hipMemcpyHostToDevice));
    }

    int k = 0;
    while (++k <= frames) {
        double t = 2*M_PI*k / frames;
        
        double cam_r = cam.r0 + cam.A*sin(cam.wr*t + cam.pr);
        double cam_z = cam.z0 + cam.Az*sin(cam.wz*t + cam.pz);
        double cam_phi = cam.phi0 + cam.wphi*t;

        double dir_r = dir.r0 + dir.A*sin(dir.wr*t + dir.pr);
        double dir_z = dir.z0 + dir.Az*sin(dir.wz*t + dir.pz);
        double dir_phi = dir.phi0 + dir.wphi*t;

        auto cam_pos = make_double3(cam_r*cos(cam_phi), cam_r*sin(cam_phi), cam_z);
        auto cam_view = make_double3(dir_r*cos(dir_phi), dir_r*sin(dir_phi), dir_z);

        hipEvent_t start, stop;
        CSC(hipEventCreate(&start));
        CSC(hipEventCreate(&stop));
        CSC(hipEventRecord(start));

        if (GPU) {
            kernel_render<<<dim3(32, 32), dim3(16, 16)>>>(dev, dev_polygons, cam_pos, cam_view, w*sqrtSamples, h*sqrtSamples, angle, lpos, lcol);
            CSC(hipGetLastError());
            
            kernel_ssaa<<<dim3(32, 32), dim3(16, 16)>>>(dev, dev_out, w, h, sqrtSamples);
            CSC(hipGetLastError());
            
            CSC(hipMemcpy(data_out, dev_out, sizeof(uchar4) *screen_size, hipMemcpyDeviceToHost));
        } else {
            make_cpu_render(data, polygons, cam_pos, cam_view, w * sqrtSamples, h * sqrtSamples, angle, lpos, lcol);
            make_cpu_ssaa(data, data_out, w, h, sqrtSamples);
        }

        CSC(hipEventRecord(stop));
        CSC(hipEventSynchronize(stop));
        
        float time;
        CSC(hipEventElapsedTime(&time, start, stop));
        CSC(hipEventDestroy(start));
        CSC(hipEventDestroy(stop));
        
        char buff[BUFF_SIZE];
        sprintf(buff, output_dir, k);
        writeToFile(buff, w, h, w*h, data_out);

        print_log(k, time, screen_size*sqrtSamples*sqrtSamples);
    }

    free(data);
    free(data_out);
    if (GPU) {
        CSC(hipFree(dev));
        CSC(hipFree(dev_out));
    }
    return 0;
}

