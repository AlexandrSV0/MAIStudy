
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>

#define CSC(call)                                                                \
    do {                                                                                 \
        hipError_t res = call;                                                 \
        if (res != hipSuccess)                                   			   \
        {                                                          					        \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n",  \
                    __FILE__, __LINE__, hipGetErrorString(res)); 	 \
            exit(0);                                              			   			  \
        }                                                        							 \
    } while (0);

const float FLOAT_MIN = -1.7976931348623158e+308;
//добавил константую память
__constant__ float3 avg_dev[32];
__constant__ float3 covInv_dev[32*3]; 
__constant__ float covDet_modif_dev[32]; 
 

// avg
__host__ void calculateAvg(float3 *avg, uchar4 *data, std::vector<std::vector<int2>> classes, int w) {
    int classCount = classes.size();
    for (int i = 0; i < classCount; i++) {
       avg[i] = make_float3(0.f, 0.f, 0.f);

        for (int j = 0; j < classes[i].size(); j++) {
            int2 coord = classes[i][j];
            uchar4 pix = data[coord.x + coord.y*w];

            avg[i].x += pix.x; 
            avg[i].y += pix.y; 
            avg[i].z += pix.z; 
        }

        int classSize = classes[i].size();
        avg[i].x /= classSize; 
        avg[i].y /= classSize;  
        avg[i].z /= classSize; 
    }
}

// cov
__host__ void calculateCov(float3 (*cov)[3], uchar4 *data, float3 *avg, std::vector<std::vector<int2>> classes, int w) {
    int classCount = classes.size();

    for (int i = 0; i < classCount; i++) {
        for (int k = 0; k < 3; k++) {
            cov[i][k] = make_float3(0.f, 0.f, 0.f);
        }

        float3 rgb_avg = avg[i];
        int classSize = classes[i].size();

        for (int j = 0; j < classSize; j++) {
            int2 coord = classes[i][j];
            uchar4 pix = data[coord.x + coord.y * w];

            for (int k = 0; k < 3; k++) {
                float diffX = pix.x - rgb_avg.x;
                float diffY = pix.y - rgb_avg.y; 
                float diffZ = pix.z - rgb_avg.z; 

                cov[i][k].x += diffX * ((k == 0) ? diffX : (k == 1) ? diffY : diffZ); 
                cov[i][k].y += diffY * ((k == 0) ? diffX : (k == 1) ? diffY : diffZ); 
                cov[i][k].z += diffZ * ((k == 0) ? diffX : (k == 1) ? diffY : diffZ); 
            }  
        }

        for (int k = 0; k < 3; k++) {
            cov[i][k].x /= classSize-1;
            cov[i][k].y /= classSize-1;
            cov[i][k].z /= classSize-1;
        }
    }
}

// det(cov)
__host__ void calculateCovDet(float *covDet, float3 (*cov)[3], int classCount) {
    for (int i = 0;  i < classCount; i++) {
        float term1 = cov[i][0].x * cov[i][1].y * cov[i][2].z;
        float term2 = cov[i][0].y * cov[i][1].z * cov[i][2].x; 
        float term3 = cov[i][0].z * cov[i][1].x * cov[i][2].y; 
        float term4 = cov[i][0].z * cov[i][1].y * cov[i][2].x;  
        float term5 = cov[i][0].y * cov[i][1].x * cov[i][2].z; 
        float term6 = cov[i][0].x * cov[i][1].z * cov[i][2].y;

        covDet[i] = term1 + term2 + term3 - term4 - term5 - term6; 
    }
}

// cov^(-1)
__host__ void calculateCovInv(float3 *covInv, float3 (*cov)[3], float *covDet, int classCount) {
    for (int i = 0; i < classCount; i++) {
        int j = i * 3;
        float det = covDet[i];

        for (int k = 0; k < 3; k++) {
            covInv[j + k].x = (k == 0) ? cov[i][1].y * cov[i][2].z - cov[i][1].z * cov[i][2].y 
                                        : (k == 1) ? -(cov[i][0].y * cov[i][2].z - cov[i][0].z * cov[i][2].y) 
                                        : cov[i][0].y * cov[i][1].z - cov[i][0].z * cov[i][1].y; 

            covInv[j + k].y = (k == 0) ? -(cov[i][1].x * cov[i][2].z - cov[i][1].z * cov[i][2].x)
                                        : (k == 1) ? cov[i][0].x * cov[i][2].z - cov[i][0].z * cov[i][2].x
                                        : -(cov[i][0].x * cov[i][1].z - cov[i][0].z * cov[i][1].x);  

            covInv[j + k].z = (k == 0) ? cov[i][1].x * cov[i][2].y - cov[i][1].y * cov[i][2].x 
                                        : (k == 1) ? -(cov[i][0].x * cov[i][2].y - cov[i][0].y * cov[i][2].x) 
                                        : cov[i][0].x * cov[i][1].y - cov[i][0].y * cov[i][1].x; 
 
            covInv[j + k].x /= det;
            covInv[j + k].y /= det;
            covInv[j + k].z /= det;
        }
    }
}

__device__ float D_f(uchar4 pix, int j) {
    int i = j/3;
    float px_avgx = pix.x - avg_dev[i].x; 
    float py_avgy = pix.y - avg_dev[i].y; 
    float pz_avgz = pix.z - avg_dev[i].z; 

    float summand_1 = (px_avgx*covInv_dev[j].x + py_avgy*covInv_dev[j+1].x + pz_avgz*covInv_dev[j+2].x)*px_avgx; 
    float summand_2 = (px_avgx*covInv_dev[j].y + py_avgy*covInv_dev[j+1].y + pz_avgz*covInv_dev[j+2].y)*py_avgy; 
    float summand_3 = (px_avgx*covInv_dev[j].z + py_avgy*covInv_dev[j+1].z + pz_avgz*covInv_dev[j+2].z)*pz_avgz; 
    
    return -(summand_1 + summand_2 + summand_3) - covDet_modif_dev[i];
}

// ММП
__device__ int predictPixelClass(int classCount, uchar4 pix) {
    float D_max = FLOAT_MIN;
    int classNum = 0;
    
    for (int i = 0; i < classCount; i++) {
        float D_cur = D_f(pix, i*3);

        if (D_cur > D_max) {
            D_max = D_cur;
            classNum = i;
        }
    }
    return classNum;
}

__global__ void kernel(uchar4 *data, int img_size, int classCount) {
    // поменял на одномерную сетку потоков здесь и в 216 строке; не обратил внимание на цель работы
    int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    int offset_x = blockDim.x * gridDim.x;

    for (int i = id_x; i < img_size; i += offset_x) {    
        data[i].w = (unsigned char) predictPixelClass(classCount, data[i]); 
    }
}

__host__ void printFloat3(float3 num) {
    printf("f3[%f | %f | %f]", num.x, num.y, num.z); 
}

int main() {
    std::string inputFile, outputFile;
    std::cin >> inputFile >> outputFile;

    int w, h;
    std::ifstream inputFS(inputFile, std::ios::in | std::ios::binary);
    inputFS.read((char *) &w, sizeof(w));
    inputFS.read((char *) &h, sizeof(h));  
    int img_size = w*h; 
    uchar4 *data = new uchar4[img_size]; 
    inputFS.read((char *) data, img_size*sizeof(uchar4));  
    inputFS.close(); 

    int classCount; 
    std::cin >> classCount;
    std::vector<std::vector<int2>> classes;

    for (int i = 0; i < classCount; i++) { 
        int classSize; 
        std::cin >> classSize; 
        std::vector<int2> curClass(classSize); 
        classes.push_back(curClass);

        for (int j = 0; j < classSize; j++) {
            int2 coord; 
            std::cin >> coord.x >> coord.y;
            classes[i][j] = coord; 
        } 
    } 

    float3 avg_host[classCount]; 
    calculateAvg(avg_host, data, classes, w); 
    
    float3 cov_host[classCount][3];
    calculateCov(cov_host, data, avg_host, classes, w); 

    float covDet_host[classCount];  
    calculateCovDet(covDet_host, cov_host, classCount);

    float covDetModified_host[classCount];
    for (int i = 0; i < classCount; i++) 
        covDetModified_host[i] = log(abs(covDet_host[i])); 

    float3 covInv_host[classCount*3];
    calculateCovInv(covInv_host, cov_host, covDet_host, classCount);
 
    uchar4 *data_dev; 
    CSC(hipMalloc(&data_dev, sizeof(uchar4)*img_size));
    CSC(hipMemcpy(data_dev, data, sizeof(uchar4) * img_size, hipMemcpyHostToDevice)); 

    CSC(hipMemcpyToSymbol(HIP_SYMBOL(avg_dev), avg_host, sizeof(float3)*classCount,0, hipMemcpyHostToDevice)); 
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(covInv_dev), covInv_host, sizeof(float3)*classCount*3,0, hipMemcpyHostToDevice)); 
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(covDet_modif_dev), covDetModified_host, sizeof(float)*classCount,0, hipMemcpyHostToDevice)); 
 
    hipEvent_t start, stop;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&stop));
    CSC(hipEventRecord(start));

    kernel<<<1024, 1024>>>(data_dev, img_size, classCount); 
    CSC(hipGetLastError());

    CSC(hipEventRecord(stop));
    CSC(hipEventSynchronize(stop));

    float t;
    CSC(hipEventElapsedTime(&t, start, stop));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(stop));

    printf("time = %f ms\n", t);

    CSC(hipMemcpy(data, data_dev, sizeof(uchar4)*img_size, hipMemcpyDeviceToHost)); 
    
    CSC(hipFree(data_dev));

	std::ofstream outputFS(outputFile, std::ios::out | std::ios::binary);  
	outputFS.write((char *) &w, sizeof(int));   
	outputFS.write((char *) &h, sizeof(int)); 
	outputFS.write((char *) data, img_size*sizeof(uchar4));  
	outputFS.close();
    free(data);
}